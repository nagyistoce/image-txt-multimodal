#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2011, Alex Krizhevsky (akrizhevsky@gmail.com)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * - Redistributions of source code must retain the above copyright notice,
 *   this list of conditions and the following disclaimer.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

//#include <cutil_inline.h>
//#include <nvmatrix.cuh>
//#include <cudaconv2.cuh>


#ifndef DIVUP
#define DIVUP(x, y) (((x) + (y) - 1) / (y))
#endif

/*
 * Block size B_YxB_X. Each block applies B_Y * filtersPerThread filters to B_X * imgsPerThread images.
 * threadIdx.x determines image
 * threadIdx.y determines filter
 *
 * blockIdx.x determines image batch of B_X * imgsPerThread
 * blockIdx.y determines filter batch of module and B_Y * filtersPerThread
 *
 * images:      (numColors, imgSizeY, imgSizeX, numImages) with stride given
 * filters:     (numColors, filterPixels, numFilters) if conv
 *              (numModules, numColors, filterPixels, numFilters) otherwise
 *
 * targets:     (numFilters, numModulesY, numModulesX, numImages)
 *
 * B_Y one of 4, 8, 16; 		filtersPerThread one of 1, 2, 4, 8
 * B_X one of 16, 32; 			imgsPerThread one of 1, 2, 4
 *
 *
 *
 * Number of filters per module should be divisible by B_Y * filtersPerThread
 * checkImgBounds indicates whether number of images is divisible by B_X * imgsPerThread
 *
 * The imgSize here is the size of the actual image without the padding.
 *
 */
template <int B_Y, int B_X, int imgsPerThread, int filtersPerThread, int numColors,
          bool scale, bool checkImgBounds, typename T>
__global__ void filterActs_YxX_color(T* images, T* filters, T* targets,
                                   const int numImages, const int numFilters,
                                   const int imgSizeY, const int imgSizeX, const int filterSize,
                                   const int paddingStart, const int moduleStride,
                                   const int numModulesY, const int numModulesX, const int imgStride,
                                   const T scaleTargets, const T scaleOutputs,
                                   const bool conv)
{
    __shared__ T shFilters[B_Y*numColors][B_Y * filtersPerThread]; // pre-load B_Y pixels from B_Y*filtersPerThread filters
    __shared__ T shImages[B_Y*numColors][B_X * imgsPerThread]; // pre-load B_Y pixels from B_X*imgsPerThread images
    const int imgPixels = imgSizeY * imgSizeX;
    const int filterPixels = filterSize * filterSize;

    const int blocksPerModule = numFilters / (B_Y*filtersPerThread);  //how many blocks are dedicated to each module(grid point)
    const int moduleIdx = blockIdx.y / blocksPerModule;			//which module (grid point)
    const int blockFilterIdx = blockIdx.y % blocksPerModule;	//which block within a module (which can have multiple blocks)

    const int tidx = threadIdx.y * B_X + threadIdx.x;	//row-major scan within each block

    const int imgLoadModPosY = (moduleIdx / numModulesX) * moduleStride; //row-major scan of modules
    const int imgLoadModPosX = (moduleIdx % numModulesX) * moduleStride;

    const int shFilterLoadY = tidx / (B_Y * filtersPerThread);			//weird re-use of threads on shared memory loading (max # of threads is 128)
    const int shFilterLoadX = tidx % (B_Y * filtersPerThread);			//maybe useful if # of threads > B_Y*filtersPerThread
    const int myImgIdx = blockIdx.x * B_X * imgsPerThread + threadIdx.x;
    images += myImgIdx;
    filters += filtersPerThread * B_Y * blockFilterIdx + shFilterLoadY * numFilters + shFilterLoadX;

    if (!conv) {
        filters += moduleIdx * numColors * filterPixels * numFilters;
    }

    targets += moduleIdx * numImages
            + (blockFilterIdx * B_Y * filtersPerThread + threadIdx.y) * numImages * numModulesY * numModulesX
            + myImgIdx;


    T prod[filtersPerThread][imgsPerThread];
    #pragma unroll
    for(int f = 0; f < filtersPerThread; f++) {
        #pragma unroll
        for(int g = 0; g < imgsPerThread; g++) {
            prod[f][g] = 0;
        }
    }

    for (int p = 0; p < filterPixels; p += B_Y) {

        //Load B_Y pixels from B_Y*filtersPerThread filters
        if (shFilterLoadY < B_Y) {
            #pragma unroll
            for (int p2 = 0; p2 < B_Y; p2 += B_X/filtersPerThread) {
                if (p + p2 + shFilterLoadY < filterPixels) {
                    #pragma unroll
                    for (int c = 0; c < numColors; c++) {
                        shFilters[shFilterLoadY + p2 + c * B_Y][shFilterLoadX] = filters[(c * filterPixels + p + p2) * numFilters];
                    }
                } else {
                    #pragma unroll
                    for (int c = 0; c < numColors; c++) {
                        shFilters[shFilterLoadY + p2 + c * B_Y][shFilterLoadX] = 0;
                    }
                }
            }
        }

        // Load B_Y pixels from B_X*imgsPerThread images
        const int pixIdx = p + threadIdx.y;
        if (pixIdx < filterPixels) {
            const int x = paddingStart + imgLoadModPosX + pixIdx % filterSize;
            const int y = paddingStart + imgLoadModPosY + pixIdx / filterSize;
            if (y >= 0 && y< imgSizeY && x >= 0 && x < imgSizeX) {
                #pragma unroll
                for (int i = 0; i < imgsPerThread; i++) {
                    if (!checkImgBounds || myImgIdx + i * B_X < numImages) {
                        #pragma unroll
                        for (int c = 0; c < numColors; c++) {
                            shImages[threadIdx.y + c * B_Y][threadIdx.x + i * B_X] = images[imgStride * (c * imgPixels + y * imgSizeX + x) + i * B_X];
                        }
                    } else {
                        #pragma unroll
                        for (int c = 0; c < numColors; c++) {
                            shImages[threadIdx.y + c * B_Y][threadIdx.x + i * B_X] = 0;
                        }
                    }
                }
            } else { // Padding
                #pragma unroll
                for (int i = 0; i < imgsPerThread; i++) {
                    #pragma unroll
                    for (int c = 0; c < numColors; c++) {
                        shImages[threadIdx.y + c * B_Y][threadIdx.x + i * B_X] = 0;
                    }
                }
            }
        }
        __syncthreads();
        #pragma unroll
        for (int i = 0; i < B_Y*numColors; i++) {
            #pragma unroll
            for(int f = 0; f < filtersPerThread; f++) {
                #pragma unroll
                for(int g = 0; g < imgsPerThread; g++) {
                    prod[f][g] += shImages[i][g * B_X + threadIdx.x] * shFilters[i][threadIdx.y + f * B_Y];
                }
            }

        }
        __syncthreads();
    }
    
    if (scale) {
        #pragma unroll
        for (int g = 0; g < imgsPerThread; g++) {
            if (!checkImgBounds || myImgIdx + g * B_X < numImages) {
                #pragma unroll
                for (int f = 0; f < filtersPerThread; f++) {
                    targets[g * B_X + f * B_Y * numImages * numModulesY * numModulesX] = scaleTargets * targets[g * B_X + f * B_Y * numImages * numModulesY * numModulesX] + scaleOutputs * prod[f][g];
                }
            }
        }
    } else {
        #pragma unroll
        for (int g = 0; g < imgsPerThread; g++) {
            if (!checkImgBounds || myImgIdx + g * B_X < numImages) {
                #pragma unroll
                for (int f = 0; f < filtersPerThread; f++) {
                    targets[g * B_X + f * B_Y * numImages * numModulesY * numModulesX] = scaleOutputs * prod[f][g];
                }
            }
        }
    }
}





/*
 * Block size B_YxB_X. Each block applies B_Y * filtersPerThread filters to B_X * imgsPerThread images.
 * threadIdx.x determines image
 * threadIdx.y determines filter
 *
 * blockIdx.x determines image batch of B_X * imgsPerThread
 * blockIdx.y determines filter batch of B_Y * filtersPerThread
 *
 * images:      (numImgColors, imgSizeY, imgSizeX, numImages) with stride given
 * filters:     (numFilterColors, filterPixels, numFilters) if conv
 *              (numModules, numFilterColors, filterPixels, numFilters) otherwise
 *
 * targets:     (numFilters, numModulesY, numModulesX, numImages)
 *
 * B_Y one of 4, 8, 16
 * B_X one of 16, 32
 * imgsPerThread one of 1, 2, 4
 * filtersPerThread one of 1, 2, 4, 8
 * colorCache: how many colors to put into shmem
 *
 * numFilters should be divisible by B_Y * filtersPerThread
 * numImages be divisible by B_X * imgsPerThread
 * numFilterColors should be divisible by colorCache.
 * numImgColors must be even.
 * numFilters must be divisible by numGroups.
 *
 * The imgSize here is the size of the actual image without the padding.
 *
 */
template <int B_Y, int B_X, int imgsPerThread, int filtersPerThread, int colorCache,
          bool scale, bool checkImgBounds, typename T>
__global__ void filterActs_YxX_sparse(T* images, T* filters, T* targets,
                                       const int numImages, const int numFilters,
                                       const int imgSizeY, const int imgSizeX, const int filterSize, const int paddingStart,
                                       const int moduleStride,
                                       const int numModulesY, const int numModulesX, const int imgStride, const int numImgColors,
                                       const int numGroups,
                                       const T scaleTargets, const T scaleOutputs,
                                       const bool conv) {
    __shared__ T shFilters[B_Y*colorCache][B_Y * filtersPerThread]; // pre-load B_Y pixels from B_Y*filtersPerThread filters
    __shared__ T shImages[B_Y*colorCache][B_X * imgsPerThread]; // pre-load B_Y pixels from B_X*imgsPerThread images
    const int imgPixels = imgSizeY * imgSizeX;
    const int filterPixels = filterSize * filterSize;
    const int numFilterColors = numImgColors / numGroups;
    const int blocksPerModule = numFilters / (B_Y*filtersPerThread);
    const int moduleIdx = blockIdx.y / blocksPerModule;
    const int blockFilterIdx = filtersPerThread * B_Y * (blockIdx.y % blocksPerModule);
    const int numFiltersPerGroup = numFilters / numGroups;
    const int blockGroupIdx = blockFilterIdx / numFiltersPerGroup;

    const int numModules = numModulesX * numModulesY;
    const int blockColorIdx = numFilterColors * blockGroupIdx;

    const int tidx = threadIdx.y * B_X + threadIdx.x;

    const int imgLoadModPosY = paddingStart + (moduleIdx / numModulesX) * moduleStride;
    const int imgLoadModPosX = paddingStart + (moduleIdx % numModulesX) * moduleStride;

    const int shFilterLoadY = tidx / (B_Y * filtersPerThread);
    const int shFilterLoadX = tidx % (B_Y * filtersPerThread);
    const int myImgIdx = blockIdx.x * B_X * imgsPerThread + threadIdx.x;

    images += blockColorIdx * imgPixels * imgStride + myImgIdx;
    filters +=blockFilterIdx
            + shFilterLoadY * numFilters + shFilterLoadX;
    if (!conv) {
        filters += moduleIdx * numFilterColors * filterPixels * numFilters;
    }

    targets += moduleIdx * numImages
            + (blockFilterIdx + threadIdx.y) * numImages * numModules
            + myImgIdx;

    T prod[filtersPerThread][imgsPerThread];
    #pragma unroll
    for(int f = 0; f < filtersPerThread; f++) {
        #pragma unroll
        for(int g = 0; g < imgsPerThread; g++) {
            prod[f][g] = 0;
        }
    }
//    __shared__ int imgPos[]
    for (int oc = 0; oc < numFilterColors; oc += colorCache) { // oc stands for outer color (loop)
        for (int p = 0; p < filterPixels; p += B_Y) {
            /*
             * Load B_Y pixels from B_Y*filtersPerThread filters
             */
            if (shFilterLoadY < B_Y) {
                #pragma unroll
                for (int p2 = 0; p2 < B_Y; p2 += B_X/filtersPerThread) {
                    if (p + p2 + shFilterLoadY < filterPixels) {
                        #pragma unroll
                        for (int c = 0; c < colorCache; c++) {
                            shFilters[shFilterLoadY + p2 + c * B_Y][shFilterLoadX] = filters[((oc+c) * filterPixels + p + p2) * numFilters];
                        }
                    } else {
                        #pragma unroll
                        for (int c = 0; c < colorCache; c++) {
                            shFilters[shFilterLoadY + p2 + c * B_Y][shFilterLoadX] = 0;
                        }
                    }
                }
            }

            /*
             * Load B_Y pixels from B_X*imgsPerThread images
             */
            const int pixIdx = p + threadIdx.y;
            if (pixIdx < filterPixels) {
                const int x = imgLoadModPosX + pixIdx % filterSize;
                const int y = imgLoadModPosY + pixIdx / filterSize;
                if (y >= 0 && y < imgSizeY && x >= 0 && x < imgSizeX) {
                    T* m = &images[imgStride * (oc * imgPixels + y * imgSizeX + x)];
                    #pragma unroll
                    for (int i = 0; i < imgsPerThread; i++) {
                        if (!checkImgBounds || myImgIdx + i * B_X < numImages) {
                            #pragma unroll
                            for (int c = 0; c < colorCache; c++) {
                                shImages[threadIdx.y + c * B_Y][threadIdx.x + i * B_X] = m[c * imgStride * imgPixels + i * B_X];
                            }
                        } else {
                            #pragma unroll
                            for (int c = 0; c < colorCache; c++) {
                                shImages[threadIdx.y + c * B_Y][threadIdx.x + i * B_X] = 0;
                            }
                        }
                    }
                } else { // Padding
                    #pragma unroll
                    for (int i = 0; i < imgsPerThread; i++) {
                        #pragma unroll
                        for (int c = 0; c < colorCache; c++) {
                            shImages[threadIdx.y + c * B_Y][threadIdx.x + i * B_X] = 0;
                        }
                    }
                }
            }
            __syncthreads();
            #pragma unroll
            for (int i = 0; i < B_Y*colorCache; i++) {
                #pragma unroll
                for(int f = 0; f < filtersPerThread; f++) {
                    #pragma unroll
                    for(int g = 0; g < imgsPerThread; g++) {
                        prod[f][g] += shImages[i][g * B_X + threadIdx.x] * shFilters[i][threadIdx.y + f * B_Y];
                    }
                }

            }
            __syncthreads();
        }
    }

    if (scale) {
        #pragma unroll
        for (int g = 0; g < imgsPerThread; g++) {
            if (!checkImgBounds || myImgIdx + g * B_X < numImages) {
                #pragma unroll
                for (int f = 0; f < filtersPerThread; f++) {
                    targets[g * B_X + f * B_Y * numImages * numModules] = scaleTargets * targets[g * B_X + f * B_Y * numImages * numModules] + scaleOutputs * prod[f][g];
                }
            }
        }
    } else {
        #pragma unroll
        for (int g = 0; g < imgsPerThread; g++) {
            if (!checkImgBounds || myImgIdx + g * B_X < numImages) {
                #pragma unroll
                for (int f = 0; f < filtersPerThread; f++) {
                    targets[g * B_X + f * B_Y * numImages * numModules] = scaleOutputs * prod[f][g];
                }
            }
        }
    }
}




/*
 * images:      (numImgColors, imgSizeY, imgSizeX; numImages) with stride given
 * 	// images does not need to be a contiguous matrix
 *
 * filters:     (numFilterColors, filterPixels; numFilters)             if conv
 *              (numModules, numFilterColors, filterPixels; numFilters) otherwise
 *
 * targets:     (numFilters, numModules; numImages)
 * 
 * Note: all of these convolution routines are optimized for the case when
 * the number of images (i.e. the minibatch size) is a multiple of 128. 
 * Other batch sizes will work, but but I made no attempt whatsoever
 * to make them work fast. 
 *
 * scaleTargets, scaleOutput:
 * targets = scaleTargets*(old targets) + scaleOutput*(images conv filter)
 * conv: true for conv wt sharing, false for local filters
 */
template<typename T>
 void _filterActs( IN const clMatrix<T>& images, int imgStride, IN const clMatrix<T>& filters, OUT clMatrix<T>& targets,
                   int imgSizeY, int numModulesY, int numModulesX, int paddingStart, int moduleStride,
                   int numImgColors, int numGroups,
                   T scaleTargets, T scaleOutput, bool conv){

    int numFilterColors = numImgColors / numGroups;
    int numFilters = filters.nI;
    int numModules = numModulesY * numModulesX;
    int numImages = images.nI;
    int imgPixels = images.nJ / numImgColors;
    int imgSizeX = imgPixels / imgSizeY;
    int filterModuleMult = conv ? 1 : numModules;
    
    clASSERT(numGroups > 1 || (numImgColors > 0 && (numImgColors <= 3 || numImgColors % 2 == 0)),"err1001");
    clASSERT(numGroups == 1 || numFilterColors % 2 == 0,"err1002");
    clASSERT(numFilters % (16 * numGroups) == 0,"err1003");
    clASSERT(numImgColors % numGroups == 0,"err1004");
    clASSERT(images.nJ == imgPixels * numImgColors,"err1005");
    clASSERT(imgSizeY * imgSizeX == imgPixels,"err1006");
    int numFiltersPerGroup = numFilters / numGroups;

    int filterPixels = filters.nJ / (filterModuleMult * numFilterColors);
    int filterSize = int(sqrt(filterPixels));
    clASSERT(filterSize * filterSize == filterPixels,"err1007");
    clASSERT(filters.nJ == filterModuleMult * numFilterColors * filterPixels,"err1008");

    // These routines don't handle the case when only part of the image is visited in the convolution
    clASSERT(paddingStart <= 0,"err1009");
    clASSERT(paddingStart + (numModulesX-1)*moduleStride + filterSize >= imgSizeX,"err1010");
    clASSERT(paddingStart + (numModulesY-1)*moduleStride + filterSize >= imgSizeY,"err1011");
    clASSERT(moduleStride <= filterSize,"err1012");
    
    //assert(!images.isTrans());
    //assert(!filters.isTrans());
    //assert(!targets.isTrans());

    //assert(filters.isContiguous());
    //assert(targets.isContiguous());

    int imgsPerThread = numImages % 128 == 0 ? 4 : numImages % 64 == 0 ? 2 : 1;
    dim3 blocks = numFiltersPerGroup % 32 == 0 ? dim3(DIVUP(numImages, 32 * imgsPerThread), (numModules * numFilters) / (4 * 8))
                                               : dim3(DIVUP(numImages, 32 * imgsPerThread), (numModules * numFilters) / (4 * 4));
    dim3 threads(32, 4);
    bool checkImgBounds = numImages % (32*imgsPerThread) != 0;
    if (scaleTargets == 0) {
        //targets.resize(numFilters * numModules, numImages);
    	clASSERT(targets.nJ == numFilters * numModules, "err1013");
        clASSERT(targets.nI == numImages, "err1014");
    } else {
        clASSERT(targets.nJ == numFilters * numModules, "err1013");
        clASSERT(targets.nI == numImages, "err1014");
    }
    
    if (imgsPerThread == 4) {
        if (numImgColors <= 3) {
            clASSERT(numGroups == 1, "err1015"); // It has to be based on above definitions, but just to be sure.
            if (scaleTargets == 0) { // don't scale
                if (numImgColors == 1) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 1, false, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 8, 1, false, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 1, false, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 4, 1, false, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 1, false, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 8, 1, false, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 1, false, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 4, 1, false, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                } else if (numImgColors == 2) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 2, false, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 8, 2, false, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 2, false, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 4, 2, false, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 2, false, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 8, 2, false, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 2, false, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 4, 2, false, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                }  else if (numImgColors == 3) {
                    if (checkImgBounds) {
                         if (numFilters % 32 == 0) {
                             hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 3, false, true, T>, hipFuncCachePreferShared);
                             filterActs_YxX_color < 4, 32, 4, 8, 3, false, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         } else {
                             hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 3, false, true, T>, hipFuncCachePreferShared);
                             filterActs_YxX_color < 4, 32, 4, 4, 3, false, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         }
                    } else {
                         if (numFilters % 32 == 0) {
                             hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 3, false, false, T>, hipFuncCachePreferShared);
                             filterActs_YxX_color < 4, 32, 4, 8, 3, false, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         } else {
                             hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 3, false, false, T>, hipFuncCachePreferShared);
                             filterActs_YxX_color < 4, 32, 4, 4, 3, false, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         }
                    }
                }
            } else { // do scale
                if (numImgColors == 1) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 1, true, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 8, 1, true, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 1, true, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 4, 1, true, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 1, true, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 8, 1, true, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 1, true, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 4, 1, true, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                } else if (numImgColors == 2) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 2, true, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 8, 2, true, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 2, true, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 4, 2, true, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 2, true, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 8, 2, true, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 2, true, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 4, 2, true, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                }  else if (numImgColors == 3) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 3, true, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 8, 3, true, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 3, true, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 4, 3, true, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 3, true, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 8, 3, true, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 3, true, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 4, 4, 3, true, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                }
            }
        } else {
            if (scaleTargets == 0) { // don't scale
                if (checkImgBounds) {
                    if (numFiltersPerGroup % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 4, 8, 2, false, true, T>, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 4, 8, 2, false, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 4, 4, 2, false, true, T>, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 4, 4, 2, false, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                } else {
                    if (numFiltersPerGroup % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 4, 8, 2, false, false, T>, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 4, 8, 2, false, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 4, 4, 2, false, false, T>, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 4, 4, 2, false, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                }
            } else { // do scale
                if (checkImgBounds) {
                    if (numFiltersPerGroup % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 4, 8, 2, false, true, T>, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 4, 8, 2, true, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 4, 4, 2, false, true, T>, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 4, 4, 2, true, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                } else {
                    if (numFiltersPerGroup % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 4, 8, 2, false, false, T>, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 4, 8, 2, true, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 4, 4, 2, false, false, T>, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 4, 4, 2, true, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                }
            }
        }
    } else if (imgsPerThread == 2) {
        if (numImgColors <= 3) {
            clASSERT(numGroups == 1, "err1016"); // It has to be based on above definitions, but just to be sure.
            if (scaleTargets == 0) { // don't scale
                if (numImgColors == 1) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 2, 8, 1, false, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 8, 1, false, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 2, 4, 1, false, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 4, 1, false, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 2, 8, 1, false, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 8, 1, false, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 2, 4, 1, false, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 4, 1, false, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                } else if (numImgColors == 2) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 2, 8, 2, false, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 8, 2, false, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 2, 4, 2, false, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 4, 2, false, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 2, 8, 2, false, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 8, 2, false, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 2, 4, 2, false, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 4, 2, false, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                }  else if (numImgColors == 3) {
                    if (checkImgBounds) {
                         if (numFilters % 32 == 0) {
                             hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 2, 8, 3, false, true, T>, hipFuncCachePreferShared);
                             filterActs_YxX_color < 4, 32, 2, 8, 3, false, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         } else {
                             hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 2, 4, 3, false, true, T>, hipFuncCachePreferShared);
                             filterActs_YxX_color < 4, 32, 2, 4, 3, false, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         }
                    } else {
                         if (numFilters % 32 == 0) {
                             hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 2, 8, 3, false, false, T>, hipFuncCachePreferShared);
                             filterActs_YxX_color < 4, 32, 2, 8, 3, false, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         } else {
                             hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 2, 4, 3, false, false, T>, hipFuncCachePreferShared);
                             filterActs_YxX_color < 4, 32, 2, 4, 3, false, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         }
                    }
                }
            } else { // do scale
                if (numImgColors == 1) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 2, 8, 1, true, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 8, 1, true, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 2, 4, 1, true, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 4, 1, true, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 2, 8, 1, true, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 8, 1, true, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 2, 4, 1, true, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 4, 1, true, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                } else if (numImgColors == 2) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 2, 8, 2, true, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 8, 2, true, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 2, 4, 2, true, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 4, 2, true, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 2, 8, 2, true, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 8, 2, true, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 2, 4, 2, true, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 4, 2, true, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                }  else if (numImgColors == 3) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 2, 8, 3, true, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 8, 3, true, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 2, 4, 3, true, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 4, 3, true, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 2, 8, 3, true, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 8, 3, true, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 2, 4, 3, true, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 2, 4, 3, true, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                }
            }
        } else {
            if (scaleTargets == 0) { // don't scale
                if (checkImgBounds) {
                    if (numFiltersPerGroup % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 2, 8, 2, false, true, T>, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 2, 8, 2, false, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 2, 4, 2, false, true, T>, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 2, 4, 2, false, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                } else {
                    if (numFiltersPerGroup % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 2, 8, 2, false, false, T>, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 2, 8, 2, false, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 2, 4, 2, false, false, T>, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 2, 4, 2, false, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                }
            } else { // do scale
                if (checkImgBounds) {
                    if (numFiltersPerGroup % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 2, 8, 2, false, true, T>, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 2, 8, 2, true, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 2, 4, 2, false, true, T>, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 2, 4, 2, true, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                } else {
                    if (numFiltersPerGroup % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 2, 8, 2, false, false, T>, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 2, 8, 2, true, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 2, 4, 2, false, false, T>, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 2, 4, 2, true, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                }
            }
        }    
    } else {
        if (numImgColors <= 3) {
            clASSERT(numGroups == 1, "err1017"); // It has to be based on above definitions, but just to be sure.
            if (scaleTargets == 0) { // don't scale
                if (numImgColors == 1) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 1, 8, 1, false, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 8, 1, false, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 1, 4, 1, false, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 4, 1, false, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 1, 8, 1, false, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 8, 1, false, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 1, 4, 1, false, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 4, 1, false, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                } else if (numImgColors == 2) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 1, 8, 2, false, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 8, 2, false, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 1, 4, 2, false, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 4, 2, false, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 1, 8, 2, false, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 8, 2, false, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 1, 4, 2, false, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 4, 2, false, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                }  else if (numImgColors == 3) {
                    if (checkImgBounds) {
                         if (numFilters % 32 == 0) {
                             hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 1, 8, 3, false, true, T>, hipFuncCachePreferShared);
                             filterActs_YxX_color < 4, 32, 1, 8, 3, false, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         } else {
                             hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 1, 4, 3, false, true, T>, hipFuncCachePreferShared);
                             filterActs_YxX_color < 4, 32, 1, 4, 3, false, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         }
                    } else {
                         if (numFilters % 32 == 0) {
                             hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 1, 8, 3, false, false, T>, hipFuncCachePreferShared);
                             filterActs_YxX_color < 4, 32, 1, 8, 3, false, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         } else {
                             hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 1, 4, 3, false, false, T>, hipFuncCachePreferShared);
                             filterActs_YxX_color < 4, 32, 1, 4, 3, false, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                         numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                         }
                    }
                }
            } else { // do scale
                if (numImgColors == 1) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 1, 8, 1, true, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 8, 1, true, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 1, 4, 1, true, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 4, 1, true, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 1, 8, 1, true, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 8, 1, true, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 1, 4, 1, true, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 4, 1, true, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                } else if (numImgColors == 2) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 1, 8, 2, true, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 8, 2, true, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 1, 4, 2, true, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 4, 2, true, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 1, 8, 2, true, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 8, 2, true, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 1, 4, 2, true, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 4, 2, true, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                }  else if (numImgColors == 3) {
                    if (checkImgBounds) {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 1, 8, 3, true, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 8, 3, true, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 1, 4, 3, true, true, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 4, 3, true, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    } else {
                        if (numFilters % 32 == 0) {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 1, 8, 3, true, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 8, 3, true, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        } else {
                            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 1, 4, 3, true, false, T>, hipFuncCachePreferShared);
                            filterActs_YxX_color < 4, 32, 1, 4, 3, true, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                        numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
                        }
                    }
                }
            }
        } else {
            if (scaleTargets == 0) { // don't scale
                if (checkImgBounds) {
                    if (numFiltersPerGroup % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 1, 8, 2, false, true, T>, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 1, 8, 2, false, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 1, 4, 2, false, true, T>, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 1, 4, 2, false, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                } else {
                    if (numFiltersPerGroup % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 1, 8, 2, false, false, T>, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 1, 8, 2, false, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 1, 4, 2, false, false, T>, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 1, 4, 2, false, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                }
            } else { // do scale
                if (checkImgBounds) {
                    if (numFiltersPerGroup % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 1, 8, 2, false, true, T>, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 1, 8, 2, true, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 1, 4, 2, false, true, T>, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 1, 4, 2, true, true, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                } else {
                    if (numFiltersPerGroup % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 1, 8, 2, false, false, T>, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 1, 8, 2, true, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 1, 4, 2, false, false, T>, hipFuncCachePreferShared);
                        filterActs_YxX_sparse < 4, 32, 1, 4, 2, true, false, T> <<<blocks, threads>>>(images.pData, filters.pData, targets.pData,
                                    numImages, numFilters, imgSizeY, imgSizeX, filterSize, paddingStart, moduleStride, numModulesY, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
                    }
                }
            }
        }
    }
    
    getLastCudaError("filterActs: kernel execution failed");
}
